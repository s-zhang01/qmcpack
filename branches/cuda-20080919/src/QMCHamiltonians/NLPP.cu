
#include <hip/hip_runtime.h>
template<typename T, int BS>
__device__
T min_dist (T x, T y, T z, 
	    T L[3][3], T Linv[3][3], T images[27][3])
{
  int tid = threadIdx.x;
  T u0 = Linv[0][0]*x + Linv[0][1]*y + Linv[0][2]*z;  
  T u1 = Linv[1][0]*x + Linv[1][1]*y + Linv[1][2]*z;
  T u2 = Linv[2][0]*x + Linv[2][1]*y + Linv[2][2]*z;

  u0 -= rintf(u0);
  u1 -= rintf(u1);
  u2 -= rintf(u2);

  x = L[0][0]*u0 + L[0][1]*u1 + L[0][2]*u2;
  y = L[1][0]*u0 + L[1][1]*u1 + L[1][2]*u2;
  z = L[2][0]*u0 + L[2][1]*u1 + L[2][2]*u2;

  __shared__ T dist2[27];
  dist2[tid] = 1.0e8;
  if (tid < 27) {
    x += images[tid][0];
    y += images[tid][1];
    z += images[tid][2];
    dist2[tid] = x*x + y*y + z*z;
  }
  __syncthreads();
  for (int s=BS>>1; s>0; s>>=1) {
    if (tid < s)
      dist2[tid] = (dist2[tid+s] < dist2[tid]) ? dist2[tid+s] : dist2[tid];
    __syncthreads();
  }

  return sqrtf(dist2[0]);
}




template<typename T, int BS>
__global__ void
find_core_electrons_kernel(T *R[], int numElec,
			   T I[], int firstIon, int lastIon,
			   T rcut, T L[3][3], T Linv[3][3],
			   int2 *pairs[], T *dist[], int numPairs[])
{
  int tid = threadIdx.x;
  __shared__ T images[27][3];
  __shared__ T *myR, *mydist;
  __shared__ int2 *mypairs;
  if (tid == 0) {
    myR     =     R[blockIdx.x];
    mydist  =  dist[blockIdx.x];
    mypairs = pairs[blockIdx.x];
  }

  int i0 = tid / 9;
  int i1 = (tid - 9*i0)/3;
  int i2 = (tid - 9*i0 - 3*i1);
  if (tid < 27) {
    images[tid][0] = (T)i0*L[0][0] + (T)i1*L[1][0] + (T)i2*L[2][0];
    images[tid][1] = (T)i0*L[0][1] + (T)i1*L[1][1] + (T)i2*L[2][1];
    images[tid][2] = (T)i0*L[0][2] + (T)i1*L[1][2] + (T)i2*L[2][2];
  }
  __syncthreads();


  int numIon = lastIon - firstIon + 1;
  int numElecBlocks = numElec/BS + ((numElec % BS) ? 1 : 0);
  int numIonBlocks  = numIon /BS + ((numIon  % BS) ? 1 : 0);

  __shared__ T r[BS][3];
  __shared__ T i[BS][3];
  __shared__ int2 blockpairs[BS];
  __shared__ T blockdist[BS];
  int npairs=0, index=0, blockNum=0;


  for (int iBlock=0; iBlock<numIonBlocks; iBlock++) {
    for (int dim=0; dim<3; dim++) 
      if (dim*BS+tid < 3*numIon)
	i[0][dim*BS+tid] = I[3*BS*iBlock + 3*firstIon + dim*BS+tid];
    int ionEnd = ((iBlock+1)*BS < numIon) ? BS : (numIon - iBlock*BS);

    for (int eBlock=0; eBlock<numElecBlocks; eBlock++) {
      int elecEnd = ((eBlock+1)*BS < numElec) ? BS : (numElec - eBlock*BS);
      for (int dim=0; dim<3; dim++) 
	if (dim*BS+tid < 3*numElec)
	  r[0][dim*BS+tid] = myR[3*BS*eBlock + dim*BS+tid];
      for (int ion=0; ion<ionEnd; ion++)
	for (int elec=0; elec<elecEnd; elec++) {
	  T dist = min_dist<T,BS>(r[elec][0]-i[ion][0], r[elec][1]-i[ion][1],
				  r[elec][2]-i[ion][2], L, Linv, images);
	  if (dist < rcut) {
	    if (index < BS) {
	      if (tid == 0) {
		blockpairs[index].x = iBlock*BS+ion;
		blockpairs[index].y = eBlock*BS+elec;
		blockdist[index]    = dist;
		index++;
	      }
	    }
	    else {
	      mypairs[blockNum*BS+tid] = blockpairs[tid];
	      mydist[blockNum*BS+tid]  = blockdist[tid];
	      blockNum++;
	      index = 0;
	    }
	    npairs++;
	  }
	}
    }
    
  }
  // Write pairs and distances remaining the final block
  if (tid < index) {
    mypairs[blockNum*BS+tid] = blockpairs[tid];
    mydist[blockNum*BS+tid]  = blockdist[tid];
  }
  if (tid == 0)
    numPairs[blockIdx.x] = npairs;
}



void
find_core_electrons (float *R[], int numElec, 
		     float I[], int firstIon, int lastIon,
		     float rcut, float L[3][3], float Linv[3][3], 
		     int2 *pairs[], float *dist[], 
		     int numPairs[], int numWalkers)
{
  const int BS = 32;
  
  dim3 dimBlock(BS);
  dim3 dimGrid(numWalkers);
  
  find_core_electrons_kernel<float,BS><<<dimGrid,dimBlock>>> 
    (R, numElec, I, firstIon, lastIon, rcut, L, Linv, pairs, dist, numPairs);
}



// Maximum quadrature points of 32;

template<typename T, int BS>
__global__ void
make_work_list_kernel (int2 *pairs[], int numpairs,
		       T I[], T quadpoints[], int numquadpoints,
		       T *ratio_pos[])
{
  __shared__ T qp[BS][3];
  
  int tid = threadIdx.x;
  for (int i=0; i<3; i++)
    if (tid*i*BS < 3*BS)
      qp[0][i*BS + tid] = quadpoints[i*BS + tid];
  __syncthreads();

  


}
